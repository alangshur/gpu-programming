
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void saxpy(int n, float a, float *x, float *y, float *out) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        out[i] = a * x[i] + y[i];
    }
}

int main(void) {
    const size_t N = 10;

    std::vector<float> x(N);
    std::vector<float> y(N);
    for (size_t i = 0; i < N; ++i) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    float *d_x, *d_y, *d_out;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));

    hipMemcpy(d_x, x.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y.data(), N * sizeof(float), hipMemcpyHostToDevice);

    saxpy<<<1, 10>>>(N, 2.0f, d_x, d_y, d_out);
    hipDeviceSynchronize();

    std::vector<float> out(N);
    hipMemcpy(out.data(), d_out, N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "out: ";
    for (float i : out) std::cout << i << " ";
    std::cout << std::endl;

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_out);

    return 0;
}