
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

__global__ void saxpy(int n, float a, float *x, float *y, float *out) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        out[i] = a * x[i] + y[i];
    }
}

int main(void) {
    const size_t N = 1000;
    const size_t NUM_THREADS = 256;

    std::vector<float> x(N);
    std::vector<float> y(N);
    for (size_t i = 0; i < N; ++i) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    float *d_x, *d_y, *d_out;
    hipMalloc(&d_x, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_out, N * sizeof(float));

    hipMemcpy(d_x, x.data(), N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y.data(), N * sizeof(float), hipMemcpyHostToDevice);

    const size_t num_blocks = (N + NUM_THREADS - 1) / NUM_THREADS;
    saxpy<<<num_blocks, NUM_THREADS>>>(N, 2.0f, d_x, d_y, d_out);
    hipDeviceSynchronize();

    std::vector<float> out(N);
    hipMemcpy(out.data(), d_out, N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "out: ";
    for (float i : out) std::cout << i << " ";
    std::cout << std::endl;

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_out);

    return 0;
}