#include "hip/hip_runtime.h"
#include <vector>
#include <algorithm>

#include "utils/cuda.cuh"
#include "mm.cuh"

#define BLOCK_DIM 32

__global__ void
mm(size_t m, size_t n, size_t p, float *x, float *y, float *out)
{
    __shared__ float x_tile[BLOCK_DIM][BLOCK_DIM];
    __shared__ float y_tile[BLOCK_DIM][BLOCK_DIM];

    size_t row = blockIdx.y * BLOCK_DIM + threadIdx.y;
    size_t col = blockIdx.x * BLOCK_DIM + threadIdx.x;

    size_t tile_row = threadIdx.y;
    size_t tile_col = threadIdx.x;

    size_t num_tiles = (n + BLOCK_DIM - 1) / BLOCK_DIM;

    float sum = 0.0;

    for (size_t tile_idx = 0; tile_idx < num_tiles; ++tile_idx)
    {
        size_t x_tile_row = blockIdx.y * BLOCK_DIM + threadIdx.y;
        size_t x_tile_col = tile_idx * BLOCK_DIM + threadIdx.x;
        if (x_tile_row < m && x_tile_col < n)
        {
            x_tile[tile_row][tile_col] = x[x_tile_row * n + x_tile_col];
        }
        else
        {
            x_tile[tile_row][tile_col] = 0.0;
        }

        size_t y_tile_row = tile_idx * BLOCK_DIM + threadIdx.y;
        size_t y_tile_col = blockIdx.x * BLOCK_DIM + threadIdx.x;
        if (y_tile_row < n && y_tile_col < p)
        {
            y_tile[tile_row][tile_col] = y[y_tile_row * p + y_tile_col];
        }
        else
        {
            y_tile[tile_row][tile_col] = 0.0;
        }

        __syncthreads();

        for (size_t k = 0; k < BLOCK_DIM; ++k)
        {
            sum += x_tile[tile_row][k] * y_tile[k][tile_col];
        }

        __syncthreads();
    }

    out[row * p + col] = sum;
}

MM::MM(const std::vector<float> &x, const std::vector<float> &y, const size_t m, const size_t n, const size_t p)
    : m_(m), n_(n), p_(p), x_(x), y_(y)
{
    // verify vector sizes
    if (x.size() != m_ * n_ || y.size() != n_ * p_)
        throw std::runtime_error("Vector sizes do not match");
    else if (x.size() == 0 || y.size() == 0)
        throw std::runtime_error("Vector size cannot be zero");

    // create stream
    CUDA_CALL(hipStreamCreate(&stream_));

    // create events
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    // allocate device memory
    CUDA_CALL(hipMallocAsync(&d_x, m_ * n_ * sizeof(float), stream_));
    CUDA_CALL(hipMallocAsync(&d_y, n_ * p_ * sizeof(float), stream_));
    CUDA_CALL(hipMallocAsync(&d_out, m_ * p_ * sizeof(float), stream_));

    // copy data to device
    CUDA_CALL(hipMemcpyAsync(d_x, x.data(), m_ * n_ * sizeof(float), hipMemcpyHostToDevice, stream_));
    CUDA_CALL(hipMemcpyAsync(d_y, y.data(), n_ * p_ * sizeof(float), hipMemcpyHostToDevice, stream_));
}

MM::~MM()
{
    hipStreamSynchronize(stream_);

    // destroy stream
    hipStreamDestroy(stream_);

    // free device memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_out);
}

void
MM::run()
{
    dim3 threads_per_block(BLOCK_DIM, BLOCK_DIM);
    dim3 blocks_per_grid((p_ + BLOCK_DIM - 1) / BLOCK_DIM, (m_ + BLOCK_DIM - 1) / BLOCK_DIM);

    // record start event
    CUDA_CALL(hipEventRecord(start, stream_));

    // launch kernel
    mm<<<blocks_per_grid, threads_per_block, 0, stream_>>>(m_, n_, p_, d_x, d_y, d_out);
    CUDA_CALL(hipGetLastError());

    // record stop event
    CUDA_CALL(hipEventRecord(stop, stream_));
}

std::vector<float>
MM::get()
{
    std::vector<float> out(m_ * p_);
    CUDA_CALL(hipMemcpyAsync(out.data(), d_out, m_ * p_ * sizeof(float), hipMemcpyDeviceToHost, stream_));
    CUDA_CALL(hipStreamSynchronize(stream_));
    return out;
}

float
MM::time()
{
    float ms;
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipEventElapsedTime(&ms, start, stop));
    return ms;
}