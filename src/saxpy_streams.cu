
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <chrono>

#define CUDA_CALL(call)                                                           \
    do                                                                            \
    {                                                                             \
        hipError_t err = call;                                                   \
        if (hipSuccess != err)                                                   \
        {                                                                         \
            std::cerr << "CUDA error in " << __FILE__ << " at line " << __LINE__; \
            std::cerr << ": " << hipGetErrorString(err) << std::endl;            \
            throw std::runtime_error("CUDA call failed");                         \
        }                                                                         \
    } while (0)

__global__ void
saxpy(int n, float a, float *x, float *y, float *out)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        out[i] = a * x[i] + y[i];
    }
}

/**
 * @class SAXPY
 * @brief A class to perform the SAXPY operation using CUDA.
 *
 * The SAXPY operation is defined as Y = a * X + Y.
 */
class SAXPY
{
public:
    /**
     * @brief Constructs a SAXPY object.
     * @param a Scalar multiplier.
     * @param x Reference to the input vector X.
     * @param y Reference to the input vector Y.
     */
    SAXPY(float a, std::vector<float> &x, std::vector<float> &y) : a_(a), x_(x), y_(y), n_(x.size())
    {
        // verify vector sizes
        if (x.size() != y.size() || x.size() == 0)
            throw std::runtime_error("Vector sizes do not match");

        // create stream
        CUDA_CALL(hipStreamCreate(&stream_));

        // allocate device memory
        CUDA_CALL(hipMallocAsync(&d_x, n_ * sizeof(float), stream_));
        CUDA_CALL(hipMallocAsync(&d_y, n_ * sizeof(float), stream_));
        CUDA_CALL(hipMallocAsync(&d_out, n_ * sizeof(float), stream_));

        // copy data to device
        CUDA_CALL(hipMemcpyAsync(d_x, x.data(), n_ * sizeof(float), hipMemcpyHostToDevice, stream_));
        CUDA_CALL(hipMemcpyAsync(d_y, y.data(), n_ * sizeof(float), hipMemcpyHostToDevice, stream_));
    }

    /**
     * @brief Destroys the SAXPY object.
     */
    ~SAXPY()
    {
        hipStreamSynchronize(stream_);

        // destroy stream
        hipStreamDestroy(stream_);

        // free device memory
        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_out);
    }

    void
    run()
    {
        const size_t num_threads = std::max<size_t>(256, n_);
        const size_t num_blocks = (n_ + num_threads - 1) / num_threads;
        saxpy<<<num_blocks, num_threads, 0, stream_>>>(n_, a_, d_x, d_y, d_out);
        CUDA_CALL(hipGetLastError());
    }

    std::vector<float>
    get()
    {
        std::vector<float> out(n_);
        CUDA_CALL(hipMemcpyAsync(out.data(), d_out, n_ * sizeof(float), hipMemcpyDeviceToHost, stream_));
        CUDA_CALL(hipStreamSynchronize(stream_));
        return out;
    }

private:
    const size_t n_;
    const float a_;
    const std::vector<float> &x_;
    const std::vector<float> &y_;

    float *d_x;
    float *d_y;
    float *d_out;

    hipStream_t stream_;
};

int
main(void)
{
    const size_t N = 1000;
    std::vector<float> x(N);
    std::vector<float> y(N);
    for (size_t i = 0; i < N; ++i)
    {
        x[i] = 2.0f;
        y[i] = 3.0f;
    }

    SAXPY saxpy1(2.0f, x, y);
    SAXPY saxpy2(3.0f, x, y);

    saxpy1.run();
    saxpy2.run();

    std::vector<float> out1 = saxpy1.get();
    std::vector<float> out2 = saxpy2.get();

    std::cout << "out1[0] = " << out1[0] << std::endl;
    std::cout << "out2[0] = " << out2[0] << std::endl;

    return 0;
}