#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>

#include "mat.cuh"

std::vector<float>
create_mat(size_t m, size_t n, float val)
{
    std::vector<float> mat(m * n);
    for (int i = 0; i < m * n; ++i) mat[i] = val;
    return mat;
}

std::vector<float>
create_random_mat(size_t m, size_t n, float min, float max)
{
    // seed random number generator
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(min, max);

    // create matrix
    std::vector<float> mat(m * n);
    for (int i = 0; i < m * n; ++i) mat[i] = dist(gen);

    return mat;
}

std::vector<float>
create_incremental_mat(size_t m, size_t n)
{
    std::vector<float> mat(m * n);
    for (int i = 0; i < m * n; ++i) mat[i] = i;
    return mat;
}

std::vector<float>
create_random_vec(size_t n, float min, float max)
{
    // seed random number generator
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(min, max);

    // create vector
    std::vector<float> vec(n);
    for (int i = 0; i < n; ++i) vec[i] = dist(gen);

    return vec;
}

void
print_mat(std::vector<float> mat, size_t m, size_t n)
{
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            std::cout << mat[i * n + j] << "\t";
        }
        std::cout << std::endl;
    }
}

void
print_vec(std::vector<float> vec, size_t n)
{
    for (int i = 0; i < n; ++i)
    {
        std::cout << vec[i] << std::endl;
    }
}