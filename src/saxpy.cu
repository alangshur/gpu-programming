#include "hip/hip_runtime.h"
#include <vector>
#include <algorithm>

#include "utils/cuda.cuh"
#include "saxpy.cuh"

__global__ void
saxpy(size_t n, float a, float *x, float *y, float *out)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        out[i] = a * x[i] + y[i];
    }
}

SAXPY::SAXPY(const float a, const std::vector<float> &x, const std::vector<float> &y, const size_t n)
    : a_(a), x_(x), y_(y), n_(n)
{
    // verify vector sizes
    if (x.size() != n_ || x.size() != y.size())
        throw std::runtime_error("Vector sizes do not match");
    else if (x.size() == 0 || y.size() == 0)
        throw std::runtime_error("Vector size cannot be zero");

    // create stream
    CUDA_CALL(hipStreamCreate(&stream_));

    // create events
    CUDA_CALL(hipEventCreate(&start));
    CUDA_CALL(hipEventCreate(&stop));

    // allocate device memory
    CUDA_CALL(hipMallocAsync(&d_x, n_ * sizeof(float), stream_));
    CUDA_CALL(hipMallocAsync(&d_y, n_ * sizeof(float), stream_));
    CUDA_CALL(hipMallocAsync(&d_out, n_ * sizeof(float), stream_));

    // copy data to device
    CUDA_CALL(hipMemcpyAsync(d_x, x.data(), n_ * sizeof(float), hipMemcpyHostToDevice, stream_));
    CUDA_CALL(hipMemcpyAsync(d_y, y.data(), n_ * sizeof(float), hipMemcpyHostToDevice, stream_));
}

SAXPY::~SAXPY()
{
    hipStreamSynchronize(stream_);

    // destroy stream
    hipStreamDestroy(stream_);

    // free device memory
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_out);
}

void
SAXPY::run()
{
    const size_t num_threads = std::min<size_t>(256, n_);
    const size_t num_blocks = (n_ + num_threads - 1) / num_threads;

    // record start event
    CUDA_CALL(hipEventRecord(start, stream_));

    // launch kernel
    saxpy<<<num_blocks, num_threads, 0, stream_>>>(n_, a_, d_x, d_y, d_out);
    CUDA_CALL(hipGetLastError());

    // record stop event
    CUDA_CALL(hipEventRecord(stop, stream_));
}

std::vector<float>
SAXPY::get()
{
    std::vector<float> out(n_);
    CUDA_CALL(hipMemcpyAsync(out.data(), d_out, n_ * sizeof(float), hipMemcpyDeviceToHost, stream_));
    CUDA_CALL(hipStreamSynchronize(stream_));
    return out;
}

float
SAXPY::time()
{
    float ms;
    CUDA_CALL(hipEventSynchronize(stop));
    CUDA_CALL(hipEventElapsedTime(&ms, start, stop));
    return ms;
}
